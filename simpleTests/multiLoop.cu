#include "hip/hip_runtime.h"
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */
__global__
void loop(int N)
{
	index = thread.Idx + block.Idx*blockDim.x
	if ( index < N)
	{
    		printf("This is iteration number %d\n", index);
    	}
  
}

int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, be sure to use more than 1 block in
   * the execution configuration.
   */

  int N = 10;
  <<<2,32>>>loop(N);
  hipDeviceSynchronize();
}
